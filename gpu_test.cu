#include <stdio.h>
#include <stdlib.h>

#ifndef __NVCC__
#warning Please compile CUDA code with CC=nvcc.
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>
#endif

#include <hipblas.h>

#include "stream.h"

static const int print_errors = 1;

static inline int cuda_check(hipError_t rc)
{
    if (rc!=hipSuccess && print_errors) {
        printf("CUDA error: %s\n", hipGetErrorString(rc));
    }
    return rc;
}

static inline int cublas_check(hipblasStatus_t rc)
{
    if (rc!=HIPBLAS_STATUS_SUCCESS && print_errors) {
        printf("CUBLAS error: %d\n", rc);
    }
    return rc;
}

int gpu_per_node()
{
    int nd;
    cuda_check( hipGetDeviceCount(&nd) );
    return nd;
}

int gpu_stream(int times, int n, float * copy, float * add, float * scale, float * triad)
{
    hipblasHandle_t handle;
    cublas_check( hipblasCreate( &handle) );

    size_t bytes = n * sizeof(double);
    double * A;
    double * B;
    double * C;
    double * R;

    cuda_check( hipMalloc((void**)&A, bytes) );
    cuda_check( hipMalloc((void**)&B, bytes) );
    cuda_check( hipMalloc((void**)&C, bytes) );
    cuda_check( hipMalloc((void**)&R, bytes) );

    double value;
    {
        value = 1.0;
        cublas_check( hipblasSetVector(n, sizeof(double), &value, 0, A, 1) );
        value = 2.0;
        cublas_check( hipblasSetVector(n, sizeof(double), &value, 0, B, 1) );
        value = 0.0;
        cublas_check( hipblasSetVector(n, sizeof(double), &value, 0, C, 1) );
    }

    for (int i=0; i<times; ++i) {

    }

    cuda_check( hipFree((void**)&A) );
    cuda_check( hipFree((void**)&B) );
    cuda_check( hipFree((void**)&C) );
    cuda_check( hipFree((void**)&R) );

    cublas_check( hipblasDestroy(handle) );

    return 0;
}
