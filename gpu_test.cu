#include <stdio.h>
#include <stdlib.h>

#in

#ifndef __NVCC__
#warning Please compile CUDA code with CC=nvcc.
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>
#endif

static const int print_errors = 1;

static inline int cuda_check(hipError_t rc)
{
    if (rc!=hipSuccess && print_errors) {
        printf("CUDA error: %s\n", hipGetErrorString(rc));
    }
    return rc;
}

int gpu_per_node()
{
    int nd;
    cuda_check( hipGetDeviceCount(&nd) );
}

int gpu_stream(size_t n, float * copy, float * add, float * scale, float * triad)
{
    size_t bytes = n * sizeof(double);
    double * A;
    double * B;
    double * C;

    cuda_check( hipMalloc((void**)&A, bytes);
    cuda_check( hipMalloc((void**)&B, bytes);
    cuda_check( hipMalloc((void**)&C, bytes);



    cuda_check( hipFree((void**)&A);
    cuda_check( hipFree((void**)&B);
    cuda_check( hipFree((void**)&C);

}
